#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on
 * performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <pthread.h>
#include <stdint.h>
#include <unistd.h>
#define MAX_LINE_LENGTH 40
#include <openssl/sha.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

#include "simpleMultiGPU.h"

#define PWD_LEN 40
    FILE *file1;
    FILE *file2;
    char pwd[sizeof(char)*(PWD_LEN + 1)];
    char *current;
////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
char *fir;
const int MAX_GPU_COUNT = 32;
const int DATA_N = 1048576 * 32;

////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA Sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadN = gridDim.x * blockDim.x;
  float sum = 0;

  for (int pos = tid; pos < N; pos += threadN) sum += d_Input[pos];

  d_Result[tid] = sum;
}
char password_good[40] = {'\0', '\0'};  //this changed only once, when we found the good passord
char password[40+1] = {'\0','\0'}; //this contains the actual password
char hfile[255];    //the hashes file name
long counter = 0;    //this couning probed passwords
int finished = 0;

void sha256(const char *input, char *output) {
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, input, strlen(input));
    SHA256_Final((unsigned char*)output, &sha256);
}

char *nextpass() {
    char line[MAX_LINE_LENGTH * sizeof(char*)];
    
    while (fgets(line, MAX_LINE_LENGTH, file2) != NULL) {
        line[strcspn(line, "\n")] = '\0';
        strcpy(pwd, line);
    }

    return pwd;
}

void status_thread() {
    int pwds;

    const short status_sleep = 1;
    while(1) {
        sleep(status_sleep);
        pwds = counter / status_sleep;
        counter = 0;

        if (finished != 0) {
            break;
        }
        
        printf("Probing: '%s' [%d pwds/sec]\n", password, pwds);
        }
}

char *crack_thread() {
    char line1[MAX_LINE_LENGTH];
    char cur[SHA256_DIGEST_LENGTH];
    char lane2[SHA256_DIGEST_LENGTH];
    char hashed_password[SHA256_DIGEST_LENGTH * 2 + 1]; // Each byte of hash produces two characters in hex
    file2 = fopen("/usr/local/share/rockyou.txt", "r");
    while (1) {
        current = nextpass();
        file1 = fopen(hfile, "r");
        while (!feof(file1)) {
            fgets(line1, MAX_LINE_LENGTH, file1);
            line1[strcspn(line1, "\n")] = '\0';
                
            sha256(current, hashed_password);
                
            for (int i = 0; i < SHA256_DIGEST_LENGTH; i++) {
                    sprintf(lane2,"%02x", (unsigned char)hashed_password[i]);
                    strcat(cur,lane2);
                }
            
            if (strcmp(cur,line1)) {
                    strcpy(password_good, current);
                    finished = 1;
                    return password_good;
                    break;
                }
        }
        
        counter++;
        
        if (finished != 0) {
            break;
        }
        
        free(current);
    }
    fclose(file1);
    fclose(file2);
    return password_good;
}


void crack_start(unsigned int threads) {
    pthread_t th[101];
    unsigned int i;

    for (i = 0; i < threads; i++) {
        (void) pthread_create(&th[i], NULL, (void *(*)(void *))crack_thread, NULL);
    }

    (void) pthread_create(&th[100], NULL, (void *(*)(void *))status_thread, NULL);

    for (i = 0; i < threads; i++) {
        (void) pthread_join(th[i], NULL);
    }

    (void) pthread_join(th[100], NULL);
}

int init(int threadsx, char *mir) {
    int threads = 1;
    threads = threadsx;
    strcpy((char*)&hfile, mir);
    crack_start(threads);
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  if (argc < 2) {
        printf("USAGE: %s hashes.ext\n",argv[0]);
	exit(1);
    }
  // Solver config
  TGPUplan plan[MAX_GPU_COUNT];

  // GPU reduction results
  float h_SumGPU[MAX_GPU_COUNT];

  float sumGPU;
  double sumCPU, diff;

  int i, j, gpuBase, GPU_N;

  const int BLOCK_N = 32;
  const int THREAD_N = 256;
  const int ACCUM_N = BLOCK_N * THREAD_N;

  printf("Starting simpleMultiGPU\n");
  checkCudaErrors(hipGetDeviceCount(&GPU_N));

  if (GPU_N > MAX_GPU_COUNT) {
    GPU_N = MAX_GPU_COUNT;
  }

  printf("CUDA-capable device count: %i\n", GPU_N);

  printf("Generating input data...\n\n");


  // Subdividing input data across GPUs
  // Get data sizes for each GPU
  for (i = 0; i < GPU_N; i++) {
    plan[i].dataN = DATA_N / GPU_N;
  }

  // Take into account "odd" data sizes
  for (i = 0; i < DATA_N % GPU_N; i++) {
    plan[i].dataN = init(100,argv[1]);
  }

  // Assign data ranges to GPUs
  gpuBase = 0;

  for (i = 0; i < GPU_N; i++) {
    plan[i].h_Sum = h_SumGPU + i;
    gpuBase += plan[i].dataN;
  }

  // Create streams for issuing GPU command asynchronously and allocate memory
  // (GPU and System page-locked)
  for (i = 0; i < GPU_N; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipStreamCreate(&plan[i].stream));
    // Allocate memory
    checkCudaErrors(
        hipMalloc((void **)&plan[i].d_Data, plan[i].dataN * sizeof(float)));
    checkCudaErrors(
        hipMalloc((void **)&plan[i].d_Sum, ACCUM_N * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Sum_from_device,
                                   ACCUM_N * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data,
                                   plan[i].dataN * sizeof(float)));

    for (j = 0; j < plan[i].dataN; j++) {
      plan[i].h_Data[j] = (float)rand() / (float)RAND_MAX;
    }
  }

  // Start timing and compute on GPU(s)
  printf("Computing with %d GPUs...\n", GPU_N);
  // create and start timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // start the timer
  sdkStartTimer(&timer);

  // Copy data to GPU, launch the kernel and copy data back. All asynchronously
  for (i = 0; i < GPU_N; i++) {
    // Set device
    checkCudaErrors(hipSetDevice(i));

    // Copy input data from CPU
    checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data,
                                    plan[i].dataN * sizeof(float),
                                    hipMemcpyHostToDevice, plan[i].stream));

    // Perform GPU computations
    reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);
    getLastCudaError("reduceKernel() execution failed.\n");

    // Read back GPU results
    checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum,
                                    ACCUM_N * sizeof(float),hipMemcpyDeviceToHost, plan[i].stream));
  }

  // Process GPU results
  for (i = 0; i < GPU_N; i++) {
    float sum;

    // Set device
    checkCudaErrors(hipSetDevice(i));

    // Wait for all operations to finish
    hipStreamSynchronize(plan[i].stream);

    // Finalize GPU reduction for current subvector
    sum = 0;

    for (j = 0; j < ACCUM_N; j++) {
      sum += plan[i].h_Sum_from_device[j];
    }

    *(plan[i].h_Sum) = (float)sum;

    // Shut down this GPU
    checkCudaErrors(hipHostFree(plan[i].h_Sum_from_device));
    checkCudaErrors(hipFree(plan[i].d_Sum));
    checkCudaErrors(hipFree(plan[i].d_Data));
    checkCudaErrors(hipStreamDestroy(plan[i].stream));
  }

  sumGPU = 0;

  for (i = 0; i < GPU_N; i++) {
    sumGPU += h_SumGPU[i];
  }

  sdkStopTimer(&timer);
  printf("  GPU Processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  // Compute on Host CPU
  printf("Computing with Host CPU...\n\n");

  sumCPU = 0;

  for (i = 0; i < GPU_N; i++) {
    for (j = 0; j < plan[i].dataN; j++) {
      sumCPU += plan[i].h_Data[j];
    }
  }

  // Compare GPU and CPU results
  printf("Comparing GPU and Host CPU results...\n");
  diff = fabs(sumCPU - sumGPU) / fabs(sumCPU);
  printf("  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU);
  printf("  Relative difference: %E \n\n", diff);


  // Cleanup and shutdown
  for (i = 0; i < GPU_N; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipHostFree(plan[i].h_Data));
  }

  return EXIT_SUCCESS;
}
